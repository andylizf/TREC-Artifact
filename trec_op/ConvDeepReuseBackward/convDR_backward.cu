#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/TensorUtils.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/detail/KernelUtils.h>
#include <c10/cuda/CUDAGuard.h>
#include <cstdint>
#include <hip/driver_types.h>

#include "convDR_backward.h"
#include "convDR_backward_kernel.cuh"

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) \
    CHECK_CUDA(x);     \
    CHECK_CONTIGUOUS(x)
[[__gnu__::__always_inline__]] inline void CHECK_SHAPE(const at::Tensor& tensor, const at::IntArrayRef shape)
{
    TORCH_CHECK(tensor.sizes() == shape, "Expected shape ", shape, " but got ", tensor.sizes());
}

struct BackwardOutput {
    at::Tensor gradInput;
    at::Tensor gradWeights;
    at::Tensor gradHash;
    at::Tensor gradBias; // Optional
};

class CovDeepReuseBackward {
private:
    at::Tensor input_row; // [n_matrices, num_rows, param_L]
    at::Tensor inputCentroids; // [n_matrices, max_buckets, param_L]
    at::Tensor weights; // [nOutputPlane, nInputPlane, kernel_height, kernel_width]
    at::Tensor gradOutput; // [batch_size, nOutputPlane, output_height, output_width]
    at::Tensor vector_index; // [n_matrices, num_rows]
    at::Tensor vector_ids; // [n_matrices, num_rows]
    at::Tensor buckets_count; // [n_matrices, max_buckets]
    at::Tensor buckets_index; // [n_matrices, total_buckets]
    at::Tensor buckets_index_inv; // [n_matrices, max_buckets]
    at::Tensor random_vectors; // [param_L, param_H]
    int64_t input_height;
    int64_t input_width;
    int64_t pad_height;
    int64_t pad_width;
    int64_t stride_height;
    int64_t stride_width;
    int64_t param_H;
    float alpha;
    float sigma;
    bool do_bias;

    int64_t n_matrices;
    int64_t param_L;
    int max_buckets;
    int nOutputPlane;
    int nInputPlane, kernel_height, kernel_width;
    int num_rows;
    int total_buckets;
    int batch_size;
    int output_height, output_width;
    at::IntArrayRef weights_sizes;

    std::pair<at::Tensor, at::Tensor> get_gradParameters(
        hipStream_t& stream,
        const at::Tensor& gradOutput_centroids // {n_matrices, max_buckets, nOutputPlane}
    ) const
    {
        at::Tensor inputCentroids_col = inputCentroids.transpose(1, 2).contiguous();
        CHECK_SHAPE(inputCentroids_col, { n_matrices, param_L, max_buckets });

        at::Tensor gradWeights = inputCentroids_col.bmm(gradOutput_centroids)
                                     .reshape({ -1, nOutputPlane })
                                     .transpose(0, 1)
                                     .reshape(weights_sizes);
        CHECK_SHAPE(gradWeights, { nOutputPlane, nInputPlane, kernel_height, kernel_width });

        at::Tensor gradBias = do_bias ? gradOutput_centroids[0].sum(0) : at::Tensor();
        return { gradWeights, gradBias };
    }

    std::pair<at::Tensor, at::Tensor> get_gradInput(
        hipStream_t& stream,
        const at::Tensor& gradOutput_centroids // {n_matrices, max_buckets, nOutputPlane}
    )
    {
        at::Tensor gradInput_rows = at::zeros({ num_rows, n_matrices * param_L }, gradOutput_centroids.options());
        at::Tensor weights_matrices = weights.reshape({ nOutputPlane, n_matrices, param_L }).transpose(0, 1).contiguous();
        at::Tensor gradInput_centroids = gradOutput_centroids.bmm(weights_matrices);
        reconstruct_gradInputRows_cuda(stream, vector_index, gradInput_centroids, gradInput_rows);

        at::Tensor gradInputs = at::zeros({ batch_size, nInputPlane, input_height, input_width },
            gradOutput_centroids.options());
        row2im_batch_cuda(stream, gradInput_rows, gradInputs,
            kernel_height, kernel_width,
            pad_height, pad_width,
            stride_height, stride_width);
        return { std::move(gradInputs), std::move(gradInput_centroids) };
    }

    at::Tensor get_gradHash(
        hipStream_t& stream,
        const at::Tensor& input_matrix,
        const at::Tensor& hash_bits,
        const at::Tensor& gradIndex)
    {
        at::Tensor grad_Hash_value = (vector_ids.unsqueeze(2).repeat({ 1, 1, max_buckets }) + 1).to(gradIndex.options()) / (buckets_index_inv.unsqueeze(1).repeat({ 1, num_rows, 1 }) + 1).to(gradIndex.options()) - 1;
        grad_Hash_value = -1 * grad_Hash_value / (sigma * sigma) * exp(-1 * grad_Hash_value * grad_Hash_value / (2 * sigma * sigma)) * gradIndex / buckets_count.unsqueeze(1).repeat({ 1, num_rows, 1 }).to(gradIndex.options());
        at::Tensor power = at::zeros({ n_matrices, max_buckets, param_H }, gradIndex.options());
        at::Tensor zero = at::zeros({ n_matrices, num_rows, max_buckets }, gradIndex.options());
        grad_Hash_value = at::where(grad_Hash_value.isnan(), zero, grad_Hash_value);

        get_Power(stream, buckets_index, power, max_buckets, param_H);
        at::Tensor gradHash = grad_Hash_value.bmm(power).reshape({ num_rows * n_matrices, param_H });

        gradHash = (alpha * hash_bits * (1 - hash_bits)) * gradHash;

        return input_matrix.transpose(0, 1).mm(gradHash);
    }

    at::Tensor get_gradOutputSum(hipStream_t& stream)
    {
        at::Tensor gradOutput_mat = gradOutput.reshape({ batch_size, nOutputPlane,
                                                           output_height * output_width })
                                        .transpose(1, 2)
                                        .reshape({ -1, nOutputPlane });
        CHECK_SHAPE(gradOutput_mat, { batch_size * output_height * output_width, nOutputPlane });
        at::Tensor gradOutput_centroids = at::zeros({ n_matrices, max_buckets, nOutputPlane }, gradOutput.options());
        get_gradOutputCentroids_add_cuda(stream, vector_index, gradOutput_mat, gradOutput_centroids);

        return gradOutput_centroids;
    }

public:
    CovDeepReuseBackward(
        const at::Tensor& input_row,
        const at::Tensor& inputCentroids,
        const at::Tensor& weights,
        const at::Tensor& gradOutput,
        const at::Tensor& vector_index,
        const at::Tensor& vector_ids,
        const at::Tensor& buckets_count,
        const at::Tensor& buckets_index,
        const at::Tensor& buckets_index_inv,
        const at::Tensor& random_vectors,
        const int64_t input_height,
        const int64_t input_width,
        const int64_t pad_height,
        const int64_t pad_width,
        const int64_t stride_height,
        const int64_t stride_width,
        const int64_t param_H,
        const float alpha,
        const float sigma,
        const bool do_bias)
        : input_row(input_row)
        , inputCentroids(inputCentroids)
        , weights(weights)
        , gradOutput(gradOutput)
        , vector_index(vector_index)
        , vector_ids(vector_ids)
        , buckets_count(buckets_count)
        , buckets_index(buckets_index)
        , buckets_index_inv(buckets_index_inv)
        , random_vectors(random_vectors)
        , input_height(input_height)
        , input_width(input_width)
        , pad_height(pad_height)
        , pad_width(pad_width)
        , stride_height(stride_height)
        , stride_width(stride_width)
        , param_H(param_H)
        , alpha(alpha)
        , sigma(sigma)
        , do_bias(do_bias)
        , n_matrices(input_row.size(0))
        , param_L(input_row.size(2))
        , max_buckets(inputCentroids.size(1))
        , nOutputPlane(weights.size(0))
        , nInputPlane(weights.size(1))
        , kernel_height(weights.size(2))
        , kernel_width(weights.size(3))
        , num_rows(vector_index.size(1))
        , total_buckets(buckets_index.size(1))
        , batch_size(gradOutput.size(0))
        , output_height(gradOutput.size(2))
        , output_width(gradOutput.size(3))
        , weights_sizes(weights.sizes())
    {
        CHECK_INPUT(input_row);
        CHECK_SHAPE(input_row, { n_matrices, num_rows, param_L });
        CHECK_INPUT(inputCentroids);
        CHECK_SHAPE(inputCentroids, { n_matrices, max_buckets, param_L });
        CHECK_INPUT(weights);
        CHECK_SHAPE(weights, { nOutputPlane, nInputPlane, kernel_height, kernel_width });
        CHECK_INPUT(gradOutput);
        CHECK_SHAPE(gradOutput, { batch_size, nOutputPlane, output_height, output_width });
        CHECK_INPUT(vector_index);
        CHECK_SHAPE(vector_index, { n_matrices, num_rows });
        CHECK_INPUT(vector_ids);
        CHECK_SHAPE(vector_ids, { n_matrices, num_rows });
        CHECK_INPUT(buckets_count);
        CHECK_SHAPE(buckets_count, { n_matrices, max_buckets });
        CHECK_INPUT(buckets_index);
        CHECK_SHAPE(buckets_index, { n_matrices, total_buckets });
        CHECK_INPUT(buckets_index_inv);
        CHECK_SHAPE(buckets_index_inv, { n_matrices, max_buckets });
        CHECK_INPUT(random_vectors);
        CHECK_SHAPE(random_vectors, { param_L, param_H });
    }

    BackwardOutput backward()
    {
        hipStream_t stream = at::cuda::getCurrentCUDAStream();

        at::Tensor gradOutput_centroids = get_gradOutputSum(stream);

        const auto& [gradWeights, gradBias] = get_gradParameters(stream, gradOutput_centroids);

        get_gradOutputCentroids_div_cuda(stream, gradOutput_centroids, buckets_count); // ? Not before get_gradParameters?

        const auto& [gradInput, gradInput_centroids] = get_gradInput(stream, gradOutput_centroids);

        at::Tensor gradIndex = input_row.bmm(gradInput_centroids.transpose(1, 2));
        at::Tensor input_matrix = input_row.reshape({ n_matrices * num_rows, param_L });
        at::Tensor hash_bits = 1 / (1 + exp(-1 * alpha * (input_matrix.mm(random_vectors) - 0.1 / pow(2, param_H))));
        const auto& gradHash = get_gradHash(stream, input_matrix, hash_bits, gradIndex);

        return { std::move(gradInput), std::move(gradWeights), std::move(gradHash), std::move(gradBias) };
    }
};

std::vector<at::Tensor> conv_deep_reuse_backward(
    const at::Tensor input_row, // {n_matrices, num_row, param_L}
    const at::Tensor inputCentroids, // {n_matrices, max_buckets, param_L}
    const at::Tensor weights, // {nOutputPlane, nInputPlane, kH, kW}
    const at::Tensor gradOutput, // {N, K, outH, outW}
    const at::Tensor vector_index, // {n_matrices, num_rows}
    const at::Tensor vector_ids, // {n_matrices, num_rows}
    const at::Tensor buckets_count, // {n_matrices, max_buckets}
    const at::Tensor buckets_index, // {n_matrices, total_buckets}
    const at::Tensor buckets_index_inv, // {n_matrices, max_buckets}
    const at::Tensor random_vectors, // {L, H}
    const int64_t input_height,
    const int64_t input_width,
    const int64_t pad_height,
    const int64_t pad_width,
    const int64_t stride_height,
    const int64_t stride_width,
    const int64_t param_H,
    const float alpha,
    const float sigma,
    const bool do_bias)
{
    auto covDeepReuseBackward = CovDeepReuseBackward {
        input_row,
        inputCentroids,
        weights,
        gradOutput,
        vector_index,
        vector_ids,
        buckets_count,
        buckets_index,
        buckets_index_inv,
        random_vectors,
        input_height,
        input_width,
        pad_height,
        pad_width,
        stride_height,
        stride_width,
        param_H,
        alpha,
        sigma,
        do_bias
    };
    auto [gradInput, gradWeights, gradHash, gradBias] = covDeepReuseBackward.backward();
    if (do_bias) {
        //! gradBias should be first to match the order in the Python code
        return { gradInput, gradWeights, gradBias, gradHash };
    }
    return { gradInput, gradWeights, gradHash };
}