#include "hip/hip_runtime.h"

#include <ATen/ATen.h>
#include <ATen/TensorUtils.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/detail/KernelUtils.h>
#include <c10/cuda/CUDACachingAllocator.h>
#include <c10/cuda/CUDAGuard.h>
#include <stdio.h>
#include <utility>

#include "convDR_forward.h"
#include "convDR_forward_kernel.cuh"
#include "func_utilis.h"

#define CHECK_CUDA(x) TORCH_CHECK(x.type().is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) \
    CHECK_CUDA(x);     \
    CHECK_CONTIGUOUS(x)

#define OUT

void LSH_projection(
    hipStream_t& stream,
    const at::Tensor& input_row,
    const at::Tensor& random_vectors, // [param_L, param_H]
    at::Tensor& vector_ids,
    at::Tensor& buckets_count,
    const int64_t param_L,
    const int64_t param_H);

void clustering_inputs(
    hipStream_t& stream,
    const at::Tensor& inputs,
    const at::Tensor& random_vectors,
    at::Tensor& vector_ids,
    at::Tensor& buckets_centroids,
    at::Tensor& buckets_count,
    at::Tensor& input_row,
    const int64_t kernel_height,
    const int64_t kernel_width,
    const int64_t pad_height,
    const int64_t pad_width,
    const int64_t stride_height,
    const int64_t stride_width,
    const int64_t param_L,
    const int64_t param_H,
    const int64_t n_matrices,
    const int64_t num_rows);

struct CentroidsInfo {
    at::Tensor centroids_for_compute;
    at::Tensor vector_index;
    at::Tensor buckets_stats;
    at::Tensor buckets_count;
    at::Tensor input_row; // only for training
    at::Tensor buckets_index; // only for training
    at::Tensor buckets_index_inv; // only for training
    at::Tensor vector_ids; // only for training
    at::Tensor remain_ratio; // only for inference
};

CentroidsInfo preprocess_inputs(
    hipStream_t& stream,
    const at::Tensor& inputs,
    const at::Tensor& random_vectors,
    const int64_t kernel_height,
    const int64_t kernel_width,
    const int64_t pad_height,
    const int64_t pad_width,
    const int64_t stride_height,
    const int64_t stride_width,
    const int64_t param_L,
    const int64_t param_H,
    bool is_training,
    bool print_rc);

std::vector<at::Tensor> conv_deep_reuse_forward(
    const at::Tensor& inputs,
    const at::Tensor& weights,
    const at::Tensor& bias,
    const at::Tensor& random_vectors,
    const int64_t pad_height,
    const int64_t pad_width,
    const int64_t stride_height,
    const int64_t stride_width,
    const int64_t param_L,
    const int64_t param_H,
    const bool do_bias,
    const bool is_training,
    const bool print_rc)
{

    // printf("conv_deep_reuse_forward\n");
    // printf("inputs.size() = %d, %d, %d, %d\n", inputs.size(0), inputs.size(1), inputs.size(2), inputs.size(3));
    // printf("weights.size() = %d, %d, %d, %d\n", weights.size(0), weights.size(1), weights.size(2), weights.size(3));
    // printf("bias.size() = %d\n", bias.size(0));
    // printf("random_vectors.size() = %d, %d\n", random_vectors.size(0), random_vectors.size(1));
    // printf("pad_height = %d, pad_width = %d, stride_height = %d, stride_width = %d\n", pad_height, pad_width, stride_height, stride_width);
    // printf("param_L = %d, param_H = %d\n", param_L, param_H);
    // printf("do_bias = %d, is_training = %d, print_rc = %d\n", do_bias, is_training, print_rc);

    CHECK_INPUT(inputs);
    CHECK_INPUT(weights);
    CHECK_INPUT(bias);
    CHECK_INPUT(random_vectors);
    // TORCH_CHECK(param_H <= 64, "paramter H must <= 64");
    TORCH_CHECK(param_H <= 32, "Paramter H must <= 32"); // hash value: int32_t

    int64_t batch_size = inputs.size(0);
    int64_t nInputPlane = inputs.size(1);
    int64_t inputHeight = inputs.size(2);
    int64_t inputWidth = inputs.size(3);

    int64_t nOutputPlane = weights.size(0);
    int64_t kernel_height = weights.size(2);
    int64_t kernel_width = weights.size(3);
    TORCH_CHECK(nInputPlane == weights.size(1), "Inconsistent number of input channels and weight channels");

    int64_t row_length = nInputPlane * kernel_width * kernel_height;
    TORCH_CHECK(row_length % param_L == 0, "Parameter L must be the factor of", row_length);
    int64_t n_matrices = row_length / param_L;

    int64_t outputHeight = (inputHeight + 2 * pad_height - kernel_height) / stride_height + 1;
    int64_t outputWidth = (inputWidth + 2 * pad_width - kernel_width) / stride_width + 1;

    hipStream_t stream = at::cuda::getCurrentCUDAStream();

    // information need for inference/training
    // val: {centroids_for_compute, vector_index, buckets_stats}
    // train: {centroids_for_compute, vector_index, buckets_stats, buckets_count_out}
    // std::vector<at::Tensor> centroids_info = preprocess_inputs(stream, inputs, random_vectors, kernel_height, kernel_width, pad_height, pad_width,
    //                                                         stride_height, stride_width, param_L, param_H, is_training, print_rc);

    // at::Tensor centroids_for_compute = centroids_info[0];
    // at::Tensor vector_index = centroids_info[1];
    // at::Tensor buckets_stats = centroids_info[2];

    const auto& [centroids_for_compute,
        vector_index,
        buckets_stats,
        buckets_count,
        input_row,
        buckets_index,
        buckets_index_inv,
        vector_ids,
        remain_ratio]
        = preprocess_inputs(stream, inputs, random_vectors,
            kernel_height, kernel_width, pad_height, pad_width, stride_height, stride_width, param_L, param_H, is_training, print_rc);

    int64_t max_buckets = buckets_stats.data_ptr<int>()[1];

    at::Tensor weights_matrices = weights.reshape({ nOutputPlane, row_length }).t().reshape({ n_matrices, param_L, nOutputPlane });

    // [n_matrices, max_buckets, n_output_plane]
    at::Tensor centroids_after_mm = centroids_for_compute.bmm(weights_matrices); // batch matrix multiplicatiion

    at::Tensor reconstructed_output = at::zeros({ batch_size, nOutputPlane, outputHeight, outputWidth }, inputs.options());
    reconstruct_output_cuda(stream, vector_index, centroids_after_mm, reconstructed_output);

    if (do_bias)
        bias_add_cuda(stream, reconstructed_output, bias);

    if (is_training) {
        return { reconstructed_output, centroids_for_compute, vector_index, vector_ids, buckets_count, buckets_index, buckets_index_inv, input_row };
    }
    return { reconstructed_output, buckets_count };
    // c10::cuda::CUDACachingAllocator::emptyCache();
    // ? Is it necessary to empty the cache?
}

void LSH_projection(
    hipStream_t& stream,
    const at::Tensor& input_row, // L sub-matrices, [n_matrices * num_rows, L]
    const at::Tensor& random_vectors, // [param_L, param_H]
    at::Tensor& vector_ids,
    at::Tensor& buckets_count,
    const int64_t param_L,
    const int64_t param_H)
{
    // at::Tensor random_vectors = at::empty({param_L, param_H}, input_row.options()).uniform_(-1, 1);
    // at::Tensor random_vectors = at::randn({param_L, param_H}, input_row.options());
    at::Tensor hashed_vectors = input_row.mm(random_vectors); // matmul -- [n_matrices * num_rows, H]
    get_id_count_cuda(stream, hashed_vectors, vector_ids, buckets_count); // compute hash value and count for each bucket
}

void clustering_inputs(
    hipStream_t& stream,
    const at::Tensor& inputs,
    const at::Tensor& random_vectors,
    at::Tensor& vector_ids,
    at::Tensor& buckets_centroids,
    at::Tensor& buckets_count,
    at::Tensor& input_row,
    const int64_t kernel_height,
    const int64_t kernel_width,
    const int64_t pad_height,
    const int64_t pad_width,
    const int64_t stride_height,
    const int64_t stride_width,
    const int64_t param_L,
    const int64_t param_H,
    const int64_t n_matrices,
    const int64_t num_rows)
{

    // real order: [n_matrices, batch_size, out_height, out_width, param_L]
    // at::Tensor input_row = at::zeros({n_matrices * num_rows, param_L}, inputs.options());
    // split the input matrix into L-submatrices(input_row)
    im2row_DRbatch_cuda(stream, inputs, input_row, kernel_height, kernel_width,
        pad_height, pad_width, stride_height, stride_width, param_L);

    LSH_projection(stream, input_row, random_vectors, vector_ids, buckets_count, param_L, param_H);

    get_centroids_add_cuda(stream, vector_ids, input_row, buckets_centroids);
}

CentroidsInfo preprocess_inputs(
    hipStream_t& stream,
    const at::Tensor& inputs,
    const at::Tensor& random_vectors,
    const int64_t kernel_height,
    const int64_t kernel_width,
    const int64_t pad_height,
    const int64_t pad_width,
    const int64_t stride_height,
    const int64_t stride_width,
    const int64_t param_L,
    const int64_t param_H,
    bool is_training,
    bool print_rc)
{
    int64_t batch_size = inputs.size(0);
    int64_t nInputPlane = inputs.size(1);
    int64_t inputHeight = inputs.size(2);
    int64_t inputWidth = inputs.size(3);
    // inputs: [batch_size, nInputPlane, inputHeight, inputWidth]

    int64_t row_length = nInputPlane * kernel_width * kernel_height;
    int64_t n_matrices = row_length / param_L; //

    int64_t outputHeight = (inputHeight + 2 * pad_height - kernel_height) / stride_height + 1;
    int64_t outputWidth = (inputWidth + 2 * pad_width - kernel_width) / stride_width + 1;
    int64_t num_rows = batch_size * outputHeight * outputWidth; //

    //* input_row,           {n_matrices, num_row, param_L}
    //* inputCentroids,      {n_matrices, max_buckets, param_L}
    //* vector_index,        {n_matrices, num_rows}
    //* vector_ids,          {n_matrices, num_rows}
    //* buckets_count,       {n_matrices, max_buckets}
    //* buckets_index,       {n_matrices, total_buckets}
    //* buckets_index_inv,   {n_matrices, max_buckets}
    //* random_vectors,      {L, H}

    at::Tensor input_row = at::zeros({ n_matrices * num_rows, param_L }, inputs.options());
    im2row_DRbatch_cuda(stream, inputs, OUT input_row, kernel_height, kernel_width,
        pad_height, pad_width, stride_height, stride_width, param_L);

    at::Tensor vector_ids = at::zeros({ n_matrices, num_rows }, inputs.options().dtype(ID_DATATYPE_AT));
    int64_t total_buckets = std::pow(2, param_H); //
    at::Tensor buckets_count = at::zeros({ n_matrices, total_buckets }, inputs.options().dtype(at::kInt));
    LSH_projection(stream, input_row, random_vectors, OUT vector_ids, OUT buckets_count, param_L, param_H);
    // * vector_ids: the bucket index of each vector (empty buckets including)

    at::Tensor buckets_centroids = at::zeros({ n_matrices, total_buckets, param_L }, inputs.options());
    get_centroids_add_cuda(stream, vector_ids, input_row, OUT buckets_centroids);
    // * the sum per element of vector in the same bucket

    input_row = input_row.reshape({ n_matrices, num_rows, param_L });

    at::Tensor buckets_index = at::zeros({ n_matrices, total_buckets }, inputs.options().dtype(at::kInt));
    at::Tensor buckets_stats = at::zeros({ 2 }, inputs.options().dtype(at::kInt));
    index_bucket_cuda(stream, buckets_count, OUT buckets_index, OUT buckets_stats);
    // * buckets_index: the uniform index of each bucket (without empty buckets)
    // * total_buckets: the total number of buckets
    // * max_buckets: the max number of buckets in each matrices

    at::Tensor vector_index = at::zeros({ n_matrices, num_rows }, inputs.options().dtype(at::kInt));
    get_vector_index_cuda(stream, vector_ids, buckets_index, OUT vector_index);
    // * vector_index: the uniform bucket index of each vector (without empty buckets)

    buckets_stats = buckets_stats.cpu();
    auto buckets_stats_ptr = buckets_stats.data_ptr<int>();
    int64_t max_buckets = buckets_stats_ptr[1];

    at::Tensor centroids_for_compute = at::zeros({ n_matrices, max_buckets, param_L }, inputs.options());
    div_remap_centroids_cuda(stream, buckets_centroids, buckets_index, buckets_count, OUT centroids_for_compute);
    // * the average per element of vector in the same bucket

    if (is_training) {
        // before: 2^H buckets
        // after: max_buckets
        // std::cout << "total_buckets=" << buckets_count.size(1) << std::endl;
        // std::cout << "max_buckets=" << max_buckets << std::endl;
        at::Tensor buckets_count_out = at::zeros({ n_matrices, max_buckets }, inputs.options().dtype(at::kInt));
        at::Tensor buckets_index_inv = at::zeros({ n_matrices, max_buckets }, inputs.options().dtype(at::kInt));
        get_buckets_count_out_cuda(stream, buckets_index, OUT buckets_index_inv, buckets_count, OUT buckets_count_out);
        // return {centroids_for_compute, vector_index, buckets_stats, buckets_count_out, input_row, buckets_index, buckets_index_inv, vector_ids};
        return {
            .centroids_for_compute = std::move(centroids_for_compute),
            .vector_index = std::move(vector_index),
            .buckets_stats = std::move(buckets_stats),
            .buckets_count = std::move(buckets_count_out),
            .input_row = std::move(input_row),
            .buckets_index = std::move(buckets_index),
            .buckets_index_inv = std::move(buckets_index_inv),
            .vector_ids = std::move(vector_ids)
        };
    }

    int64_t num_vectors = num_rows * n_matrices;
    int64_t sum_buckets = buckets_stats_ptr[0];
    double rc = (double)sum_buckets / (double)num_vectors;
    at::Tensor remain_ratio = at::tensor({ rc }, inputs.options());

    return {
        .centroids_for_compute = std::move(centroids_for_compute),
        .vector_index = std::move(vector_index),
        .buckets_stats = std::move(buckets_stats),
        .remain_ratio = at::tensor({ rc }, inputs.options())
    };
}